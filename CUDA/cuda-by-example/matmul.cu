/*
 * @FileName   : basis.cu 
 * @CreateAt   : 2021/12/8
 * @Author     : Inno Fang
 * @Email      : innofang@yeah.net
 * @Description: 
 */

#include <hip/hip_runtime.h>
#include <cstdio>

struct Matrix {
    size_t width;
    size_t height;
    int *value;
};

__device__
int getValue( Matrix *matrix, size_t row, size_t col ) {
    return matrix->value[row * matrix->width + col];
}

__device__
void setValue( Matrix *matrix, size_t row, size_t col, int val ) {
    matrix->value[row * matrix->width + col] = val;
}

__global__
void matmul( Matrix *A, Matrix *B, Matrix *C ) {
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;

    int val = 0;
    for ( size_t i = 0; i < A->width; ++ i ) {
        val += getValue(A, row, i) * getValue(B, i, col);
    }
    setValue( C, row, col, val );
}

int main() {

    Matrix *A, *B, *C;

    hipMallocManaged( (void**)&A,  sizeof(Matrix) );
    hipMallocManaged( (void**)&B,  sizeof(Matrix) );
    hipMallocManaged( (void**)&C,  sizeof(Matrix) );

    size_t width = 1 << 5;
    size_t height = 1 << 5;

    A->width = width; A->height = height;
    B->width = width; B->height = height;
    C->width = width; C->height = height;

    size_t nBytes = width * height * sizeof(size_t);
    hipMallocManaged( (void**)&A->value, nBytes );
    hipMallocManaged( (void**)&B->value, nBytes );
    hipMallocManaged( (void**)&C->value, nBytes );

    for ( int i = 0; i < width * height; ++ i ) {
        A->value[i] = 2;
        B->value[i] = 3;
    }

    dim3 blockSize( width, height );
    dim3 gridSize( (width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y );

    matmul<<<gridSize, blockSize>>>( A, B, C );

    hipDeviceSynchronize();

    for ( int i = 1; i <= width * height; ++ i ) {
        printf("%d ", C->value[i]);
        if ( i % width == 0 ) printf("\n");
    }
    printf("\n");

    hipFree( A->value );
    hipFree( B->value );
    hipFree( C->value );
    hipFree( A );
    hipFree( B );
    hipFree( C );

    return 0;
}