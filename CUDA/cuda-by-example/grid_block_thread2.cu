/*
 * @FileName   : grid_block_thread2.cu
 * @CreateAt   : 2021/12/10
 * @Author     : Inno Fang
 * @Email      : innofang@yeah.net
 * @Description: implement the way of getting the thread id in vary
 *               the dimension of the grid and the dimension of the block
 */


#include <hip/hip_runtime.h>
#include <cstdio>

#define BID_IN_GRID1 blockIdx.x
#define BID_IN_GRID2 (BID_IN_GRID1 + blockIdx.y * gridDim.x)
#define BID_IN_GRID3 (BID_IN_GRID2 + blockIdx.z * gridDim.x * gridDim.y)

#define TID_IN_BLOCK1(BLOCK_ID) (threadIdx.x \
                                    + BLOCK_ID * blockDim.x)
#define TID_IN_BLOCK2(BLOCK_ID) (threadIdx.x \
                                    + threadIdx.y * blockDim.x \
                                    + BLOCK_ID * blockDim.x * blockDim.y * blockDim.z);
#define TID_IN_BLOCK3(BLOCK_ID) (threadIdx.x \
                                    + threadIdx.y * blockDim.x \
                                    + threadIdx.z * blockDim.x * blockDim.y \
                                    + BLOCK_ID * blockDim.x * blockDim.y * blockDim.z)

__global__
void kernel() {
    size_t tid = TID_IN_BLOCK3(BID_IN_GRID2);
    printf("[log] thread idx: %lld\n", tid);
}

int main() {
    dim3 block_num(1, 2);
    dim3 thread_num(2, 3, 2);
    kernel<<<block_num, thread_num>>>();
    return 0;
}