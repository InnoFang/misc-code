/*
 * @FileName   : matadd.cu
 * @CreateAt   : 2021/12/11
 * @Author     : Inno Fang
 * @Email      : innofang@yeah.net
 * @Description: implement matrix addition in CUDA
 */


#include <hip/hip_runtime.h>
#include <cstdio>

static void HandleError(hipError_t err, const char *file, int line) {
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

#define N 64

__global__
void gpu(int *a, int *b, int *c_gpu) {
    size_t r = threadIdx.x + blockIdx.x * blockDim.x;
    size_t c = threadIdx.y + blockIdx.y * blockDim.y;

    if (r < N && c < N) {
        c_gpu[r * N + c] = a[r * N + c] + b[r * N + c];
    }
}

void cpu(int *a, int *b, int *c_cpu) {
    for (int r = 0; r < N; ++ r) {
       for (int c = 0; c < N; ++ c) {
           c_cpu[r * N + c] = a[r * N + c] + b[r * N + c];
       }
    }
}

bool check(int *c_cpu, int *c_gpu) {
    for (int r = 0; r < N; ++ r) {
        for (int c = 0; c < N; ++ c) {
            if (c_cpu[r * N + c] != c_gpu[r * N + c]) {
                return false;
            }
        }
    }
    return true;
}

int main() {
    int *a, *b, *c_cpu, *c_gpu;
    size_t size = N * N * sizeof(int);

    HANDLE_ERROR( hipMallocManaged((void**)&a, size) );
    HANDLE_ERROR( hipMallocManaged((void**)&b, size) );
    HANDLE_ERROR( hipMallocManaged((void**)&c_cpu, size) );
    HANDLE_ERROR( hipMallocManaged((void**)&c_gpu, size) );

    /* initialize */
    for (int r = 0; r < N; ++ r) {
        for (int c = 0; c < N; ++ c) {
            a[r * N + c] = r;
            a[r * N + c] = c;
            c_cpu[r * N + c] = 0;
            c_gpu[r * N + c] = 0;
        }
    }

    dim3 threads(16, 16, 1);
    dim3 blocks((N + threads.x - 1) / threads.x, (N + threads.y - 1) / threads.y, 1);

    gpu<<<blocks, threads>>>(a, b, c_gpu);
    hipDeviceSynchronize();
    cpu(a, b, c_cpu);

    check(c_cpu, c_gpu) ? printf("passed\n") : printf("failed\n");

    HANDLE_ERROR( hipFree(a) );
    HANDLE_ERROR( hipFree(b) );
    HANDLE_ERROR( hipFree(c_cpu) );
    HANDLE_ERROR( hipFree(c_gpu) );

    return 0;
}

