#include "hip/hip_runtime.h"
/*
 * @FileName   : single_stream.cu 
 * @CreateAt   : 2021/12/12
 * @Author     : Inno Fang
 * @Email      : innofang@yeah.net
 * @Description: Use single stream to calculate (A + B)/2
 */
#include <cstdio>

#define N (1024 * 1024) //每个流执行数据大小
#define FULL (N * 20)   // 全部数据大小

__global__
void kernel(int *a, int *b, int *c) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if ( idx < N ) {
        c[idx] = (a[idx] + b[idx]) >> 1;
    }
}

int main() {
    // 查询设备属性
    hipDeviceProp_t prop;
    int whichDevice;
    hipGetDevice(&whichDevice);
    hipGetDeviceProperties(&prop, whichDevice);
    if (!prop.deviceOverlap) {
        printf("Device will not support overlap\n");
        exit(1);
    }

    // 初始化计时器事件
    hipEvent_t start, stop;
    float elapsedTime;
    hipStream_t stream;
    int *host_a, *host_b, *host_c;
    int *dev_a, *dev_b, *dev_c;

    // 创建计时器
    hipEventCreate(&start);
    hipEventCreate(&stop);

    //初始化流
    hipStreamCreate(&stream);

    // 在 GPU 端申请存储空间
    hipMalloc( (void**)&dev_a, N * sizeof(int) );
    hipMalloc( (void**)&dev_b, N * sizeof(int) );
    hipMalloc( (void**)&dev_c, N * sizeof(int) );

    // 在 CPU 端申请存储空间，使用锁页内存
    hipHostAlloc( (void**)&host_a, FULL * sizeof(int), hipHostMallocDefault);
    hipHostAlloc( (void**)&host_b, FULL * sizeof(int), hipHostMallocDefault);
    hipHostAlloc( (void**)&host_c, FULL * sizeof(int), hipHostMallocDefault);

    // 初始化 A，B 向量
    for (int i = 0; i < FULL; ++ i) {
        host_a[i] = rand();
        host_b[i] = rand();
    }

    // 开始计算
    hipEventRecord(start, 0);
    for (int i = 0; i < FULL; i += N) {
        // 将数据从 CPU 锁页内存中传输给 GPU 显存
        hipMemcpyAsync(dev_a, host_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(dev_b, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream);
        kernel<<< N + 256 - 1 / 256, 256, 0, stream>>>(dev_a, dev_b, dev_c);

        // 将计算结果从 GPU 显存中传输给 CPU 内存
        hipMemcpyAsync(host_c + i, dev_c, N * sizeof(int), hipMemcpyDeviceToHost, stream);
    }
    hipStreamSynchronize(stream);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    printf("Time: %3.1f ms\n", elapsedTime);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    hipHostFree(host_a);
    hipHostFree(host_b);
    hipHostFree(host_c);
    hipStreamDestroy(stream);
    return 0;
}
