/*
 * @FileName   : grid_block_thread.cu
 * @CreateAt   : 2021/12/10
 * @Author     : Inno Fang
 * @Email      : innofang@yeah.net
 * @Description: learn how to get thread id in vary
 *               the dimension of the grid and the dimension of the block
 */


#include <hip/hip_runtime.h>
#include <cstdio>

__device__
size_t getDim1BlockId() {
    return blockIdx.x;
}

__device__
size_t getDim2BlockId() {
    return blockIdx.x
           + blockIdx.y * gridDim.x;
}

__device__
size_t getDim3BlockId() {
    return blockIdx.x
           + blockIdx.y * gridDim.x
           + blockIdx.z * gridDim.x * gridDim.y;
}

__device__
size_t getDim1ThreadId(size_t block_id) {
    return threadIdx.x
           + block_id * blockDim.x;
}

__device__
size_t getDim2ThreadId(size_t block_id) {
    return threadIdx.x
           + threadIdx.y * blockDim.x
           + block_id * blockDim.x * blockDim.y * blockDim.z;
}

__device__
size_t getDim3ThreadId(size_t block_id) {
    return threadIdx.x
           + threadIdx.y * blockDim.x
           + threadIdx.z * blockDim.x * blockDim.y
           + block_id * blockDim.x * blockDim.y * blockDim.z;
}

__device__
size_t getTheadId(size_t grid_dim = 1, size_t block_dim = 1) {
    if (grid_dim == 1 && block_dim == 1) return getDim1ThreadId(getDim1BlockId());
    if (grid_dim == 1 && block_dim == 2) return getDim1ThreadId(getDim2BlockId());
    if (grid_dim == 1 && block_dim == 3) return getDim1ThreadId(getDim3BlockId());
    if (grid_dim == 2 && block_dim == 1) return getDim2ThreadId(getDim1BlockId());
    if (grid_dim == 2 && block_dim == 2) return getDim2ThreadId(getDim2BlockId());
    if (grid_dim == 2 && block_dim == 3) return getDim2ThreadId(getDim3BlockId());
    if (grid_dim == 3 && block_dim == 1) return getDim3ThreadId(getDim1BlockId());
    if (grid_dim == 3 && block_dim == 2) return getDim3ThreadId(getDim2BlockId());
    return getDim3ThreadId(getDim3BlockId());
}

__global__
void kernel() {
    size_t tid = getTheadId();
    printf("[log] thread idx: %lld\n", tid);
}

int main() {
    kernel<<<2, 3>>>();
    return 0;
}