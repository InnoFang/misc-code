/*
 * @FileName   : matrix_transpose.cu 
 * @CreateAt   : 2021/12/19
 * @Author     : Inno Fang
 * @Email      : innofang@yeah.net
 * @Description: matrix transpose
 */


#include <hip/hip_runtime.h>
#include <cstdio>

#define CHECK(err)                                \
do {                                              \
    if (err != hipSuccess) {                     \
        printf("CUDA Error:\n");                  \
        printf("    File:       %s\n", __FILE__); \
        printf("    Line:       %d\n", __LINE__); \
        printf("    Error code: %d\n", err);      \
        printf("    Error text: %s\n",            \
                hipGetErrorString(err));         \
        exit(1);                                  \
    }                                             \
} while (0)

#ifdef USE_DP
typedef double real;
#else
typedef float real;
#endif

#define NUM_REPEATS 10
#define TILE_DIM 32
#define N 10000
#define N2 (N * N)

__global__
void transpose1(const real *A, real *B) {
    const unsigned int nx = threadIdx.x + blockIdx.x * blockDim.x;
    const unsigned int ny = threadIdx.y + blockIdx.y * blockDim.y;
    if (nx < N && ny < N) {
        // READ coalesced WRITE uncoalesced
        B[nx * N + ny] = A[ny * N + nx];
    }
}

__global__
void transpose2(const real *A, real *B) {
    const unsigned int nx = threadIdx.x + blockIdx.x * blockDim.x;
    const unsigned int ny = threadIdx.y + blockIdx.y * blockDim.y;
    if (nx < N && ny < N) {
        // READ uncoalesced WRITE coalesced
        B[ny * N + nx] = A[nx * N + ny];
    }
}

__global__
void transpose3(const real *A, real *B) {
    const unsigned int nx = threadIdx.x + blockIdx.x * blockDim.x;
    const unsigned int ny = threadIdx.y + blockIdx.y * blockDim.y;
    if (nx < N && ny < N) {
        // READ uncoalesced WRITE coalesced
        B[ny * N + nx] = __ldg(&A[nx * N + ny]);
    }
}

__global__
void transpose4(const real *A, real *B) {
    __shared__ real S[TILE_DIM][TILE_DIM];
    unsigned int bx = blockIdx.x * TILE_DIM;
    unsigned int by = blockIdx.y * TILE_DIM;

    unsigned int nx1 = bx + threadIdx.x;
    unsigned int ny1 = by + threadIdx.y;
    if (nx1 < N && ny1 < N) {
        S[threadIdx.y][threadIdx.x] = A[ny1 * N + nx1];
    }
    __syncthreads();

    unsigned int nx2 = bx +threadIdx.x;
    unsigned int ny2 = by +threadIdx.y;
    if (nx2 < N && ny2 < N) {
        B[nx2 * N + ny2] = S[threadIdx.x][threadIdx.y];
    }
}

__global__
void transpose5(const real *A, real *B) {
    __shared__ real S[TILE_DIM][TILE_DIM + 1];
    unsigned int bx = blockIdx.x * TILE_DIM;
    unsigned int by = blockIdx.y * TILE_DIM;

    unsigned int nx1 = bx + threadIdx.x;
    unsigned int ny1 = by + threadIdx.y;
    if (nx1 < N && ny1 < N) {
        S[threadIdx.y][threadIdx.x] = A[ny1 * N + nx1];
    }
    __syncthreads();

    unsigned int nx2 = bx +threadIdx.x;
    unsigned int ny2 = by +threadIdx.y;
    if (nx2 < N && ny2 < N) {
        B[nx2 * N + ny2] = S[threadIdx.x][threadIdx.y];
    }
}

void timing(const real *d_A, real *d_B, const int task) {
    const unsigned grid_size_x = (N + TILE_DIM - 1) / TILE_DIM;
    const unsigned grid_size_y = grid_size_x;
    const dim3 block_size(TILE_DIM, TILE_DIM);
    const dim3 grid_size(grid_size_x, grid_size_y);

    float t_sum = 0;
    float t2_sum = 0;
    for (int repeat = 0; repeat <= NUM_REPEATS; ++repeat) {
        hipEvent_t start, stop;
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));
        CHECK(hipEventRecord(start));
        hipEventQuery(start);

        if (task == 1) transpose1<<<grid_size, block_size>>>(d_A, d_B);
        else if (task == 2) transpose2<<<grid_size, block_size>>>(d_A, d_B);
        else if (task == 3) transpose3<<<grid_size, block_size>>>(d_A, d_B);
        else if (task == 4) transpose4<<<grid_size, block_size>>>(d_A, d_B);
        else if (task == 5) transpose5<<<grid_size, block_size>>>(d_A, d_B);
        else {
            printf("Error: wrong task\n");
            exit(1);
        }

        CHECK(hipEventRecord(stop));
        CHECK(hipEventSynchronize(stop));

        float elapsed_time;
        CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
        printf("Time = %g ms.\n", elapsed_time);

        if (repeat > 0) {
            t_sum += elapsed_time;
            t2_sum += elapsed_time * elapsed_time;
        }

        CHECK(hipEventDestroy(start));
        CHECK(hipEventDestroy(stop));
    }

    const float t_ave = t_sum / NUM_REPEATS;
    const float t_err = sqrt(t2_sum / NUM_REPEATS - t_ave * t_ave);
    printf("Time = %g +- %g ms.", t_ave, t_err);
}

void print_matrix(const real *A) {
    for (int ny = 0; ny < N; ny++) {
        for (int nx = 0; nx < N; nx++) {
            printf("%g\t", A[ny * N + nx]);
        }
        printf("\n");
    }
}

int main() {
    const int M = sizeof(real) * N2;
    real *h_A = (real *) malloc(M);
    real *h_B = (real *) malloc(M);
    for (int n = 0; n < N2; ++n) {
        h_A[n] = n;
    }
    real *d_A, *d_B;
    CHECK(hipMalloc((void **) &d_A, M));
    CHECK(hipMalloc((void **) &d_B, M));
    CHECK(hipMemcpy(d_A, h_A, M, hipMemcpyHostToDevice));

    printf("\ntranspose with coalesced read:\n");
    timing(d_A, d_B, 1);
    printf("\ntranspose with coalesced write:\n");
    timing(d_A, d_B, 2);
    printf("\ntranspose with coalesced write and __ldg read:\n");
    timing(d_A, d_B, 3);
    printf("\ntranspose with shared memory bank conflict:\n");
    timing(d_A, d_B, 4);
    printf("\ntranspose without shared memory bank conflict:\n");
    timing(d_A, d_B, 5);

    CHECK(hipMemcpy(h_B, d_B, M, hipMemcpyDeviceToHost));
    if (N <= 10) {
        printf("A =\n");
        print_matrix(h_A);
        printf("B =\n");
        print_matrix(h_B);
    }

    free(h_A);
    free(h_B);
    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    return 0;
}