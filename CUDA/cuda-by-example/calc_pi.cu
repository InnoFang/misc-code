/*
 * @FileName   : basis.cu 
 * @CreateAt   : 2021/12/6
 * @Author     : Inno Fang
 * @Email      : innofang@yeah.net
 * @Description: 
 */

#include <hip/hip_runtime.h>
#include <iostream>
#include <ctime>

#define N 1000000
#define real float
#define CUDA_KERNEL_LOOP(i, n) \
        for (int i = blockIdx.x * blockDim.x + threadIdx.x; \
            i < (n);           \
            i += blockDim.x * gridDim.x)

static void HandleError(hipError_t err, const char *file=__FILE__, int line=__LINE__) {
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

__global__
void distance(real *x, real *y, real *result) {
    CUDA_KERNEL_LOOP(idx, N) {
        real temp = (x[idx] - 1) * (x[idx] - 1) + (y[idx] - 1) * (y[idx] - 1);
        result[idx] = temp < 1 ? 1.0 : 0;
    }
}

__global__
void sum(real *a, real *b) {
    __shared__ real sData[512];
    b[0] = 0;
    size_t tid = threadIdx.x;
    int boundary = static_cast<int>(ceilf(N / 512));
    for (int count = 0; count < boundary; ++count) {
        if (tid + count * 512 < N) {
            sData[tid] = a[tid + count * 512];
            __syncthreads();
        }
        for (int i = 256; i > 0; i >>= 1) {
            if (tid < i && tid + count * 512 < N) {
                sData[tid] = sData[tid] + sData[tid + i];
            }
            __syncthreads();
        }
        if (tid == 0) {
            b[0] += sData[0];
        }
    }
}

int main() {
    srand(time(nullptr));
    real *x_square = new float[N];
    real *y_square = new float[N];

    for (int i = 0; i < N; ++ i) {
        x_square[i] = static_cast<real>(rand() % 10000 * 1.0 / 10000);
        y_square[i] = static_cast<real>(rand() % 10000 * 1.0 / 10000);
    }

    real *x_square_d, *y_square_d;
    HANDLE_ERROR( hipMalloc( (void**)&x_square_d, N * sizeof(real) ) );
    HANDLE_ERROR( hipMalloc( (void**)&y_square_d, N * sizeof(real) ) );
    HANDLE_ERROR( hipMemcpy(x_square_d, x_square, N * sizeof(real), hipMemcpyHostToDevice) );
    HANDLE_ERROR( hipMemcpy(y_square_d, y_square, N * sizeof(real), hipMemcpyHostToDevice) );

    int block_num = 512;
    int thread_num = 1024;
    real *result_d;
    HANDLE_ERROR( hipMalloc( (void**)&result_d, N * sizeof(real) ) );
    distance<<<block_num, thread_num>>>(x_square_d, y_square_d, result_d);

    real *result = new real[N];
    HANDLE_ERROR( hipMemcpy(result, result_d, N * sizeof(real), hipMemcpyDeviceToHost) );

    real *b_d;
    HANDLE_ERROR( hipMalloc( (void**)&b_d, 1 * sizeof(real) ) );
    sum<<<1, 512, 512>>>(result_d, b_d);

    real b[1];
    HANDLE_ERROR( hipMemcpy(b, b_d, 1 * sizeof(real), hipMemcpyDeviceToHost) );
    printf("b: %f\n", b[0]);
    printf("pi: %f\n", b[0] * 4.0 / N);

    hipFree(x_square_d);
    hipFree(y_square_d);
    hipFree(result_d);
    hipFree(b_d);

    free(x_square);
    free(y_square);
    free(result);
    return 0;
}

