#include "hip/hip_runtime.h"
/*
 * @FileName   : basis.cu 
 * @CreateAt   : 2021/12/6
 * @Author     : Inno Fang
 * @Email      : innofang@yeah.net
 * @Description: 
 */
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define N (1024 * 1024)
#define FULL (N * 20)

/* (A + B) / 2 = C */
__global__
void kernel(int *a, int *b, int *c) {
   int index = threadIdx.x + blockDim.x * blockIdx.x;
   if (index < N) {
       c[index] = (a[index] + b[index]) >> 1;
   }
}

int main() {
    hipDeviceProp_t prop;
    int whichDevice;

    // 获取设备编号
    hipGetDevice(&whichDevice);
    // 获取设备属性
    hipGetDeviceProperties(&prop, whichDevice);

    if (!prop.deviceOverlap) { // 判断GPU是否支持多流计算
        std::cout << "Device will not support overlap" << std::endl;
    }

    // 初始化计时器事件
    hipEvent_t start, stop;
    float elapsedTime;

    // 声明流和GPU Buffer的指针
    hipStream_t stream0, stream1;
    int *host_a, *host_b, *host_c;
    int *dev_a0, *dev_b0, *dev_c0,
    int *dev_a1, *dev_b1, *dev_c1;

    // 创建计时器
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // 初始化流
    hipStreamCreate(&stream0);
    hipStreamCreate(&stream1);

    // 在GPU端申请存储空间
    hipMalloc( (void**)&dev_a0, N * sizeof(int) );
    hipMalloc( (void**)&dev_b0, N * sizeof(int) );
    hipMalloc( (void**)&dev_c0, N * sizeof(int) );
    hipMalloc( (void**)&dev_a1, N * sizeof(int) );
    hipMalloc( (void**)&dev_b1, N * sizeof(int) );
    hipMalloc( (void**)&dev_c1, N * sizeof(int) );

    // 在CPU端申请内存空间，要用到锁页内存(page-locked memory)
    hipHostAlloc( (void**)&host_a, FULL * sizeof(int), hipHostMallocDefault);
    hipHostAlloc( (void**)&host_b, FULL * sizeof(int), hipHostMallocDefault);
    hipHostAlloc( (void**)&host_c, FULL * sizeof(int), hipHostMallocDefault);

    // 初始化A、B向量
    for (size_t i = 0; i < FULL; ++i) {
        host_a[i] = rand();
        host_b[i] = rand();
    }

    // 开始计算
    hipEventRecord(start, 0);
    for (size_t i= 0; i < FULL; i += 2 * N) {
        // 将数据从CPU内存中传输到GPU显存中
        hipMemcpyAsync(dev_a0, host_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream0);
        hipMemcpyAsync(dev_b0, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream0);
        hipMemcpyAsync(dev_a1, host_a + i + N, N * sizeof(int), hipMemcpyHostToDevice, stream1);
        hipMemcpyAsync(dev_b1, host_b + i + N, N * sizeof(int), hipMemcpyHostToDevice, stream1);

        kernel<<<N / 256, 256, 0, stream0>>>(dev_a0, dev_b0, dev_c0);
        kernel<<<N / 256, 256, 0, stream1>>>(dev_a1, dev_b1, dev_c1);

        hipMemcpyAsync(host_c + i, dev_c0, N * sizeof(int), hipMemcpyDeviceToHost, stream0);
        hipMemcpyAsync(host_c + i + N, dev_c1, N * sizeof(int), hipMemcpyDeviceToHost, stream1);
    }

    hipStreamSynchronize(stream0);
    hipStreamSynchronize(stream1);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Time: %3.1f ms\n", elapsedTime);

    hipFree(dev_a0);
    hipFree(dev_b0);
    hipFree(dev_c0);
    hipFree(dev_a1);
    hipFree(dev_b1);
    hipFree(dev_c1);

    hipHostFree(host_a);
    hipHostFree(host_b);
    hipHostFree(host_c);

    hipStreamDestroy(stream0);
    hipStreamDestroy(stream1);

    return 0;
}

