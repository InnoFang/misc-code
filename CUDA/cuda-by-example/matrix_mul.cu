/*
 * @FileName   : matmul.cu
 * @CreateAt   : 2021/12/8
 * @Author     : Inno Fang
 * @Email      : innofang@yeah.net
 * @Description: implement matrix multiply in CUDA
 */

#include <hip/hip_runtime.h>
#include <cstdio>
#include <cmath>

#define HANDLE_ERROR(err)                         \
do {                                              \
    if (err != hipSuccess) {                     \
        printf("CUDA Error:\n");                  \
        printf("    File:       %s\n", __FILE__); \
        printf("    Line:       %d\n", __LINE__); \
        printf("    Error code: %d\n", err);      \
        printf("    Error text: %s\n",            \
                hipGetErrorString(err));         \
        exit(1);                                  \
    }                                             \
} while (0)

struct Matrix {
    size_t width;
    size_t height;
    int *value;
};

__global__
void gpu_mat_mul(Matrix *A, Matrix *B, Matrix *C) {
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;

    int value = 0;
    if (col < A->width && row < B->height) {
        for (size_t k = 0; k < C->width; ++k) {
            value += A->value[row * A->width + k] * B->value[k * B->height + col];
        }
        C->value[row * C->width + col] = value;
    }
}

void cpu_mat_mul(Matrix *A, Matrix *B, Matrix *C) {
    for (int row = 0; row < B->height; ++row) {
        for (int col = 0; col < A->width; ++col) {
            int value = 0;
            for (int k = 0; k < C->width; ++k) {
                value += A->value[row * A->width + k] * B->value[k * B->height + col];
            }
            C->value[row * C->width + col] = value;
        }
    }
}

bool check(Matrix *A, Matrix *B) {
    for (int row = 0; row < A->height; ++row) {
        for (int col = 0; col < A->width; ++col) {
            if (fabs(A->value[row * A->width + col] - B->value[row * B->width + col]) > (1.0e-10)) {
                return false;
            }
        }
    }
    return true;
}

int main() {

    Matrix *hostA, *hostB, *hostC, *hostCC;
    Matrix *devA, *devB, *devC;

    size_t height = 1 << 5;
    size_t width = 1 << 5;
    size_t nBytes = width * height * sizeof(size_t);
    HANDLE_ERROR(hipHostMalloc((void **) &hostA, sizeof(Matrix), hipHostMallocDefault, hipHostMallocDefault, hipHostMallocDefault));
    HANDLE_ERROR(hipHostMalloc((void **) &hostB, sizeof(Matrix), hipHostMallocDefault, hipHostMallocDefault, hipHostMallocDefault));
    HANDLE_ERROR(hipHostMalloc((void **) &hostC, sizeof(Matrix), hipHostMallocDefault, hipHostMallocDefault, hipHostMallocDefault));
    HANDLE_ERROR(hipHostMalloc((void **) &hostCC, sizeof(Matrix), hipHostMallocDefault, hipHostMallocDefault, hipHostMallocDefault));

    hostA->width = width; hostA->height = height;
    hostB->width = width; hostB->height = height;
    hostC->width = width; hostC->height = height;
    hostCC->width = width; hostCC->height = height;
    HANDLE_ERROR(hipHostMalloc((void **) &hostA->value, nBytes, hipHostMallocDefault, hipHostMallocDefault, hipHostMallocDefault));
    HANDLE_ERROR(hipHostMalloc((void **) &hostB->value, nBytes, hipHostMallocDefault, hipHostMallocDefault, hipHostMallocDefault));
    HANDLE_ERROR(hipHostMalloc((void **) &hostC->value, nBytes, hipHostMallocDefault, hipHostMallocDefault, hipHostMallocDefault));
    HANDLE_ERROR(hipHostMalloc((void **) &hostCC->value, nBytes, hipHostMallocDefault, hipHostMallocDefault, hipHostMallocDefault));

    for (int i = 0; i < height; ++i) {
        for (int j = 0; j < width; ++j) {
            hostA->value[i * width + j] = rand() % 1024 + 1;
        }
    }

    for (int i = 0; i < height; ++i) {
        for (int j = 0; j < width; ++j) {
            hostB->value[i * width + j] = rand() % 1024 + 1;
        }
    }

    HANDLE_ERROR(hipMalloc((void **) &devA, sizeof(Matrix)));
    HANDLE_ERROR(hipMalloc((void **) &devB, sizeof(Matrix)));
    HANDLE_ERROR(hipMalloc((void **) &devC, sizeof(Matrix)));
    HANDLE_ERROR(hipMemcpy(devA, hostA, sizeof(Matrix), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(devB, hostB, sizeof(Matrix), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(devC, hostC, sizeof(Matrix), hipMemcpyHostToDevice));

    dim3 blockSize(width, height);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    hipEvent_t start, stop_cpu, stop_gpu;
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop_gpu));
    HANDLE_ERROR(hipEventCreate(&stop_cpu));

    HANDLE_ERROR(hipEventRecord(start));
    gpu_mat_mul<<<gridSize, blockSize>>>(devA, devB, devC);
    HANDLE_ERROR(hipMemcpy(hostC, devC, sizeof(Matrix), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipGetLastError());
    HANDLE_ERROR(hipEventRecord(stop_gpu));
    HANDLE_ERROR(hipEventSynchronize(stop_gpu));

    cpu_mat_mul(hostA, hostB, hostCC);
    HANDLE_ERROR(hipEventRecord(stop_cpu));
    HANDLE_ERROR(hipEventSynchronize(stop_cpu));

    float elapsed_time_cpu, elapsed_time_gpu;
    HANDLE_ERROR(hipEventElapsedTime(&elapsed_time_gpu, start, stop_gpu));
    HANDLE_ERROR(hipEventElapsedTime(&elapsed_time_cpu, stop_gpu, stop_cpu));
    printf("GPU Time = %g ms.\n", elapsed_time_gpu);
    printf("CPU Time = %g ms.\n", elapsed_time_cpu);

    check(hostC, hostCC) ? printf("passed") : printf("failed");

    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(stop_cpu));
    HANDLE_ERROR(hipEventDestroy(stop_gpu));

    HANDLE_ERROR(hipFree(devA));
    HANDLE_ERROR(hipFree(devB));
    HANDLE_ERROR(hipFree(devC));

    HANDLE_ERROR(hipHostFree(hostA->value));
    HANDLE_ERROR(hipHostFree(hostB->value));
    HANDLE_ERROR(hipHostFree(hostC->value));
    HANDLE_ERROR(hipHostFree(hostA));
    HANDLE_ERROR(hipHostFree(hostB));
    HANDLE_ERROR(hipHostFree(hostC));

    return 0;
}