/*
 * @FileName   : conv.cu
 * @CreateAt   : 2021/12/6
 * @Author     : Inno Fang
 * @Email      : innofang@yeah.net
 * @Description: implement basic convolution in CUDA
 */
#include <iostream>

#include <hip/hip_runtime.h>


#define WIDTH 1920
#define HEIGHT 1080
#define real float

static void HandleError(hipError_t err, const char *file, int line) {
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

int getThreadNum() {
    hipDeviceProp_t prop;
    int count;

    HANDLE_ERROR( hipGetDeviceCount(&count) );
    printf("GPU device count is %d\n", count);
    HANDLE_ERROR( hipGetDeviceProperties(&prop, 0) );
    printf("Warp size is %d\n", prop.warpSize);
    printf("Max thread number per block is %d\n", prop.maxThreadsPerBlock);
    printf("Max block number per MultiProcessor is %d\n", prop.maxBlocksPerMultiProcessor);
    printf("The dimensions of grid is (%d, %d, %d)\n",
           prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    return prop.maxThreadsPerBlock;
}

__global__
void conv(real *img, real *kernel, real *result, int kernel_dim) {
    int id = threadIdx.x + (blockIdx.x * blockDim.x);
    if (id < WIDTH * HEIGHT) {
        int row = id / WIDTH;
        int col = id % WIDTH;
        for (int i = 0; i < kernel_dim; ++ i) {
            for (int j = 0; j < kernel_dim; ++ j) {
                real img_val = 0;
                int curr_row = row - kernel_dim / 2 + i;
                int curr_col = col - kernel_dim / 2 + j;
                if (0 <= curr_row && curr_row < HEIGHT && 0 <= curr_col && curr_col < WIDTH) {
                    img_val = img[curr_row * WIDTH + curr_col];
                }
                result[id] += kernel[i * kernel_dim + j] * img_val;
            }
        }
    }
}

int main() {

    /* INITIALIZE */

    real *img = new real[WIDTH * HEIGHT];
    for (int row = 0; row < HEIGHT; ++ row) {
        for (size_t col = 0; col < WIDTH; ++ col) {
            img[col + row * WIDTH] = static_cast<real>((col + row) % 256);
        }
    }

    int kernel_dim = 3;
    int kernel_size = kernel_dim * kernel_dim;
    real *kernel = new real[kernel_size];
    for (int i = 0; i < kernel_size; ++i) {
        kernel[i] = static_cast<real>(i % kernel_dim - 1); // each row is [-1 0 1]
    }


    real *img_d, *kernel_d, *result_d;
    HANDLE_ERROR( hipMalloc((void**)&img_d, WIDTH * HEIGHT * sizeof(real)) );
    HANDLE_ERROR( hipMalloc((void**)&kernel_d, kernel_size * sizeof(real)) );
    HANDLE_ERROR( hipMalloc((void**)&result_d, WIDTH * HEIGHT * sizeof(real)) );

    HANDLE_ERROR( hipMemcpy(img_d, img, WIDTH * HEIGHT * sizeof(real), hipMemcpyHostToDevice) );
    HANDLE_ERROR( hipMemcpy(kernel_d, kernel, kernel_size * sizeof(real), hipMemcpyHostToDevice) );

    /* COMPUTING */

    int thread_num = getThreadNum();
    int block_num = ( WIDTH * HEIGHT + thread_num - 1 ) / thread_num;
    conv<<<block_num, thread_num>>>(img_d, kernel_d, result_d, kernel_dim);

    real *result = new real[WIDTH * HEIGHT];
    HANDLE_ERROR( hipMemcpy(result, result_d, WIDTH * HEIGHT * sizeof(real), hipMemcpyDeviceToHost) );

    /* DISPLAY */

    // print image
    printf("\nimage\n===========\n");
    for (int row = 0; row < 10; ++ row) {
        for (int col = 0; col < 10; ++ col) {
            printf("%3.0f", img[col + row * WIDTH]);
        }
        printf("\n");
    }

    // print kernel
    printf("\nkernel\n===========\n");
    for (int row = 0; row < kernel_dim; ++ row) {
        for (int col = 0; col < kernel_dim; ++ col) {
            printf("%3.0f", kernel[col + row * kernel_dim]);
        }
        printf("\n");
    }

    // print result
    printf("\nresult\n===========\n");
    for (size_t row = 0; row < 10; ++ row) {
        for (size_t col = 0; col < 10; ++ col) {
            printf("%3.0f", result[col + row * WIDTH]);
        }
        printf("\n");
    }

    hipFree(img_d);
    hipFree(kernel_d);
    hipFree(result_d);

    free(img);
    free(kernel);
    free(result);

    return 0;
}

