#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/fill.h>

#include <iostream>

int main () {
    /**
     * Thrust support "raw" pointer used as an argument to a Thrust function.
     * But if the pointer in question is in fact a pointer to device memory,
     * it will need to be wraped with `thrust::device_ptr` before calling the function
     */

    size_t N = 10;

    // raw pointer to device memory
    int* raw_ptr;
    hipMalloc( (void**) &raw_ptr, N * sizeof(int) );

    // wrap raw pointer with a device_ptr
    thrust::device_ptr<int> dev_ptr(raw_ptr);

    // use device_ptr in thrust algorithm
    thrust::fill(dev_ptr, dev_ptr + N, (int) 1);

    for (int i = 0; i < N; ++ i) {
        std::cout << "dev_ptr[" << i << "] = " << dev_ptr[i] << std::endl;
    }
    return 0;
}